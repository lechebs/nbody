#include "hip/hip_runtime.h"
#include "cuda/octree.cuh"

#include "cuda/soa_vec3.cuh"
#include "cuda/soa_btree_nodes.cuh"
#include "cuda/soa_octree_nodes.cuh"
#include "cuda/btree.cuh"
#include "cuda/points.cuh"
#include "cuda/utils.cuh"

#include <iostream>
#include <cmath>

// For 3 levels DFS traversal
#define _BUILD_STACK_SIZE 4

__device__ __forceinline__ int _traverse(const int *child,
                                         const int *edge_delta,
                                         int start_node)
{
    int bin_node = start_node;

    do {
        bin_node = child[bin_node];
    } while (!edge_delta[bin_node]);

    return bin_node;
}

template<typename T>
__global__ void _build_octree(const SoABtreeNodes btree_nodes,
                              SoAOctreeNodes octree_nodes,
                              T *octree_nodes_size,
                              const int *btree_octree_map,
                              const int *btree_num_leaves,
                              float domain_size,
                              int *octree_num_nodes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int btree_num_nodes = 2 * *btree_num_leaves - 1;

    if (idx >= btree_num_nodes || !btree_nodes.edge_delta(idx)) {
        return;
    }

    if (idx == 0) {
        *octree_num_nodes =
            btree_octree_map[btree_num_nodes - 1] +
            btree_nodes.edge_delta(btree_num_nodes - 1);
        //printf("num_octree_nodes=%d\n", *octree_num_nodes);
    }

    int parent = btree_octree_map[idx];

    // Resetting number of children
    octree_nodes.num_children(parent) = 0;

    int node_level = btree_nodes.lcp(idx) / 3;
    // Computing the side length of the cube spanned by the octree node
    octree_nodes_size[parent] = domain_size / (1 << node_level);

    int node_leaves_begin = btree_nodes.leaves_begin(idx);
    int node_leaves_end = btree_nodes.leaves_end(idx);

    octree_nodes.leaves_begin(parent) = node_leaves_begin;
    octree_nodes.leaves_end(parent) = node_leaves_end;

    octree_nodes.depth(parent) = btree_nodes.depth(idx);

    if (node_leaves_begin == node_leaves_end) {
        // Leaf octree node
        return;
    }

    int first_bin_child = _traverse(btree_nodes.left(),
                                    btree_nodes.edge_delta(),
                                    idx);

    int last_bin_child = _traverse(btree_nodes.right(),
                                   btree_nodes.edge_delta(),
                                   idx);

    int first_child = btree_octree_map[first_bin_child];
    int last_child = btree_octree_map[last_bin_child];

    octree_nodes.first_child(parent) = first_child;
    octree_nodes.num_children(parent) = last_child - first_child + 1;
}

__global__ void
_compute_octree_nodes_points_range(const int *leaf_first_code_idx,
                                   const int *codes_first_point_idx,
                                   const int *leaves_begin_,
                                   const int *leaves_end_,
                                   int *points_begin,
                                   int *points_end,
                                   const int *num_nodes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *num_nodes) {
        return;
    }

    int leaves_begin = leaves_begin_[idx];
    int leaves_end = leaves_end_[idx];

    int codes_begin = leaf_first_code_idx[leaves_begin];
    int codes_end = leaf_first_code_idx[leaves_end + 1] - 1;

    points_begin[idx] = codes_first_point_idx[codes_begin];
    points_end[idx] = codes_first_point_idx[codes_end + 1] - 1;
}

template<typename T> __global__ void
_compute_octree_nodes_barycenter(const SoAVec3<T> weighted_pos,
                                 const SoAVec3<T> scan_weighted_pos,
                                 const T *points_mass,
                                 const T *scan_points_mass,
                                 const int *points_begin,
                                 const int *points_end,
                                 SoAVec3<T> barycenters,
                                 T *nodes_mass,
                                 const int *num_nodes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *num_nodes) {
        return;
    }

    int begin = points_begin[idx];
    int end = points_end[idx];

    T x_barycenter = scan_weighted_pos.x(end) -
                     scan_weighted_pos.x(begin) +
                     weighted_pos.x(end);

    T y_barycenter = scan_weighted_pos.y(end) -
                     scan_weighted_pos.y(begin) +
                     weighted_pos.y(end);

    T z_barycenter = scan_weighted_pos.z(end) -
                     scan_weighted_pos.z(begin) +
                     weighted_pos.z(end);

    T mass_sum = scan_points_mass[end] -
                 scan_points_mass[begin] +
                 points_mass[end];

    nodes_mass[idx] = mass_sum;

    x_barycenter /= mass_sum;
    y_barycenter /= mass_sum;
    z_barycenter /= mass_sum;

    barycenters.x(idx) = x_barycenter;
    barycenters.y(idx) = y_barycenter;
    barycenters.z(idx) = z_barycenter;
}

template<typename T> Octree<T>::Octree(int max_num_leaves,
                                       float domain_size) :
    _domain_size(domain_size),
    _gl_buffers(false)
{
    _init(max_num_leaves);

    _barycenters.alloc(_max_num_nodes);

    hipMalloc(&_nodes_size, _max_num_nodes * sizeof(T));
}

template<typename T> Octree<T>::Octree(int max_num_leaves,
                                       float domain_size,
                                       T *d_barycenters_x,
                                       T *d_barycenters_y,
                                       T *d_barycenters_z,
                                       T *d_nodes_size) :
    _domain_size(domain_size),
    _gl_buffers(true)
{
    _init(max_num_leaves);

    _barycenters.x() = d_barycenters_x;
    _barycenters.y() = d_barycenters_y;
    _barycenters.z() = d_barycenters_z;

    _nodes_size = d_nodes_size;
}

template<typename T> void Octree<T>::_init(int max_num_leaves)
{
    _max_num_nodes = min(
        2 * max_num_leaves,
        geometric_sum(8, ceil(log2(max_num_leaves) / 3.0) + 1.0));

    hipMalloc(&_num_nodes, sizeof(int));
    hipMalloc(&_points_begin, _max_num_nodes * sizeof(int));
    hipMalloc(&_points_end, _max_num_nodes * sizeof(int));

    _nodes.alloc(_max_num_nodes);
    hipMalloc(&_nodes_mass, _max_num_nodes * sizeof(T));
}

template<typename T> void Octree<T>::build(const Btree &btree)
{
    _build_octree<<<btree.get_max_num_nodes() / THREADS_PER_BLOCK +
                    (btree.get_max_num_nodes() % THREADS_PER_BLOCK > 0),
                    THREADS_PER_BLOCK>>>(btree.get_d_nodes(),
                                         _nodes,
                                         _nodes_size,
                                         btree.get_d_octree_map_ptr(),
                                         btree.get_d_num_leaves_ptr(),
                                         _domain_size,
                                         _num_nodes);
}

template<typename T>
void Octree<T>::compute_nodes_points_range(const int *d_leaf_first_code_idx,
                                           const int *d_codes_first_point_idx)
{
    _compute_octree_nodes_points_range<<<
        _max_num_nodes / THREADS_PER_BLOCK +
        (_max_num_nodes % THREADS_PER_BLOCK > 0),
        THREADS_PER_BLOCK>>>(d_leaf_first_code_idx,
                             d_codes_first_point_idx,
                             _nodes._leaves_begin,
                             _nodes._leaves_end,
                             _points_begin,
                             _points_end,
                             _num_nodes);
}

template<typename T> __global__
void compute_octree_leaves_weighted_pos(const SoAOctreeNodes nodes,
                                        const SoAVec3<T> points_pos,
                                        const T *points_mass,
                                        const int *points_begin,
                                        const int *points_end,
                                        SoAVec3<T> barycenters,
                                        T *mass,
                                        int *num_nodes)
{
    int idx = blockIdx.x;
    if (idx >= *num_nodes) {
        return;
    }

    int num_children = nodes.num_children(idx);
    if (num_children > 0) {
        return;
    }

    int begin = points_begin[idx];
    int end = points_end[idx];

    int num_points = end - begin + 1;

    T weighted_x = 0.0;
    T weighted_y = 0.0;
    T weighted_z = 0.0;
    T tot_mass = 0.0;

    for (int i = 0; i + threadIdx.x < num_points; i += 32) {
        T m = points_mass[begin + i + threadIdx.x];

        T x = points_pos.x(begin + i + threadIdx.x) * m;
        T y = points_pos.y(begin + i + threadIdx.x) * m;
        T z = points_pos.z(begin + i + threadIdx.x) * m;

        // Warp scan
        #pragma unroll 5
        for (int delta = 16; delta > 0; delta >>= 1) {
            x += __shfl_down_sync(0xffffffff, x, delta);
            y += __shfl_down_sync(0xffffffff, y, delta);
            z += __shfl_down_sync(0xffffffff, z, delta);
            m += __shfl_down_sync(0xffffffff, m, delta);
        }

        weighted_x += x;
        weighted_y += y;
        weighted_z += z;
        tot_mass += m;
    }

    if (threadIdx.x == 0) {
        barycenters.x(idx) = weighted_x;
        barycenters.y(idx) = weighted_y;
        barycenters.z(idx) = weighted_z;
        mass[idx] = tot_mass;
    }
}

template<typename T>
__global__ void compute_octree_nodes_weighted_pos(const SoAOctreeNodes nodes,
                                                  SoAVec3<T> barycenters,
                                                  T *mass,
                                                  int *num_nodes,
                                                  int curr_depth)
{
    int node_idx = (blockDim.x * blockIdx.x + threadIdx.x) / 8;
    if (node_idx >= *num_nodes) {
        return;
    }

    int num_children = nodes.num_children(node_idx);
    int depth = nodes.depth(node_idx);
    if (num_children == 0 || depth != curr_depth) {
        return;
    }

    int first_child = nodes.first_child(node_idx);

    T m = 0.0;
    T wx = 0.0;
    T wy = 0.0;
    T wz = 0.0;

    int lane_idx = threadIdx.x % 8;
    if (lane_idx < num_children) {
        int children = first_child + lane_idx;
        m = mass[children];
        wx = barycenters.x(children);
        wy = barycenters.y(children);
        wz = barycenters.z(children);
    }

    #pragma unroll 3
    for (int delta = 4; delta > 0; delta >>= 1) {
        m += __shfl_down_sync(0xffffffff, m, delta, 8);
        wx += __shfl_down_sync(0xffffffff, wx, delta, 8);
        wy += __shfl_down_sync(0xffffffff, wy, delta, 8);
        wz += __shfl_down_sync(0xffffffff, wz, delta, 8);
    }

    if (lane_idx == 0) {
        mass[node_idx] = m;
        barycenters.x(node_idx) = wx;
        barycenters.y(node_idx) = wy;
        barycenters.z(node_idx) = wz;
    }
}

template<typename T>
__global__ void compute_octree_nodes_barycenter(SoAVec3<T> barycenters,
                                                T *mass,
                                                int *num_nodes)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= *num_nodes) {
        return;
    }

    T m = mass[idx];
    T wx = barycenters.x(idx);
    T wy = barycenters.y(idx);
    T wz = barycenters.z(idx);

    barycenters.x(idx) = wx / m;
    barycenters.y(idx) = wy / m;
    barycenters.z(idx) = wz / m;
}

template<typename T>
void Octree<T>::compute_nodes_barycenter(const Points<T> &points)
{
    /*
    _compute_octree_nodes_barycenter<<<
        _max_num_nodes / THREADS_PER_BLOCK +
        (_max_num_nodes % THREADS_PER_BLOCK > 0),
        THREADS_PER_BLOCK>>>(points.get_d_weighted_pos(),
                             points.get_d_scan_weighted_pos(),
                             points.get_d_mass(),
                             points.get_d_scan_mass(),
                             _points_begin,
                             _points_end,
                             _barycenters,
                             _nodes_mass,
                             _num_nodes);
    */

    compute_octree_leaves_weighted_pos<<<
        _max_num_nodes, 32>>>(_nodes,
                              points.get_d_pos(),
                              points.get_d_mass(),
                              _points_begin,
                              _points_end,
                              _barycenters,
                              _nodes_mass,
                              _num_nodes);

    int max_depth;
    hipMemcpy(&max_depth,
               _nodes._depth + get_num_nodes() - 1,
               sizeof(int),
               hipMemcpyDeviceToHost);

    for (int d = max_depth - 1; d >= 0; --d) {
        compute_octree_nodes_weighted_pos<<<
            (_max_num_nodes * 8 - 1) / MAX_THREADS_PER_BLOCK + 1,
            MAX_THREADS_PER_BLOCK>>>(_nodes,
                                     _barycenters,
                                     _nodes_mass,
                                     _num_nodes,
                                     d);
    }

    compute_octree_nodes_barycenter<<<
        (_max_num_nodes - 1) / MAX_THREADS_PER_BLOCK + 1,
        MAX_THREADS_PER_BLOCK>>>(_barycenters,
                                 _nodes_mass,
                                 _num_nodes);
}

template<typename T> Octree<T>::~Octree()
{
    hipFree(_num_nodes);
    hipFree(_points_begin);
    hipFree(_points_end);

    _nodes.free();

    if (!_gl_buffers) {
        _barycenters.free();
        hipFree(_nodes_size);
    }

    hipFree(_nodes_mass);
}

// Explicit templates instantiation

template class Octree<float>;
template class Octree<double>;
