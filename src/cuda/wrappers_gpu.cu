#include "CUDAWrappers.hpp"

#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "cuda/points_gpu.cuh"
#include "cuda/btree_gpu.cuh"
#include "cuda/octree_gpu.cuh"

namespace CUDAWrappers
{
    struct BarnesHut::Impl
    {
        Impl(int num_points) :
            points(num_points),
            btree(num_points),
            octree(num_points) {}

        Impl(int num_points, void *mapped_ptrs[5]) :
            points(num_points,
                   static_cast<float *>(mapped_ptrs[0]),
                   static_cast<float *>(mapped_ptrs[1]),
                   static_cast<float *>(mapped_ptrs[2])),
            btree(num_points),
            octree(num_points,
                   static_cast<int *>(mapped_ptrs[3]),
                   static_cast<int *>(mapped_ptrs[4])) {}

        void updatePoints()
        {
            points.compute_morton_codes();
            points.sort_by_codes();
            points.compute_unique_codes(btree.get_d_num_leaves_ptr());
            points.scan_attributes();
        }

        void updateOctree(int max_num_codes_per_leaf)
        {
            std::cout << "num_unique_codes=" << btree.get_num_leaves() << std::endl;
            btree.generate_leaves(points.get_d_unique_codes_ptr(),
                                  max_num_codes_per_leaf);
            btree.set_max_num_leaves(btree.get_num_leaves());
            std::cout << "num_leaves=" << btree.get_num_leaves() << std::endl;

            btree.build(points.get_d_unique_codes_ptr());
            btree.sort_to_bfs_order();
            btree.compute_octree_map();

            octree.set_max_num_nodes(btree.get_max_num_nodes());
            octree.build(btree);
            octree.compute_nodes_points_range(
                btree.get_d_leaf_first_code_idx_ptr(),
                points.get_d_codes_first_point_idx_ptr());
            octree.compute_nodes_barycenter(points);

            octree.print();
        }

        Points<float> points;
        Btree btree;
        Octree<float> octree;
    };

    BarnesHut::BarnesHut(BarnesHut::Params &params) :
        _params(params)
    {
        _impl = std::make_unique<BarnesHut::Impl>(params.num_points);
    }

    BarnesHut::BarnesHut(BarnesHut::Params &params, GLuint buffers[5]) :
        _params(params)
    {
        void *mapped_ptrs[5];

        for (int i = 0; i < 5; ++i) {
            hipGraphicsResource_t res;
            // Mapping OpenGL buffer for access by CUDA
            hipGraphicsGLRegisterBuffer(&res,
                                         buffers[i],
                                         hipGraphicsRegisterFlagsNone);
            hipGraphicsMapResources(1, &res);
            // Obtaining device pointer
            size_t size;
            hipGraphicsResourceGetMappedPointer(&mapped_ptrs[i], &size, res);
        }

        _impl = std::make_unique<BarnesHut::Impl>(params.num_points,
                                                  mapped_ptrs);
    }

    void BarnesHut::samplePoints()
    {
        _impl->points.sample_uniform();
    }

    void BarnesHut::update()
    {
        _impl->updatePoints();
        _impl->updateOctree(_params.max_num_codes_per_leaf);
    }

    BarnesHut::~BarnesHut() {}
};
