#include "CUDAWrappers.hpp"

#include <string>

#include <GL/glew.h>
#include <cuda_gl_interop.h>

#include "cuda/points.cuh"
#include "cuda/btree.cuh"
#include "cuda/octree.cuh"
#include "cuda/barnes_hut.cuh"
#include "cuda/validator.cuh"
#include "cuda/spawner.cuh"

namespace CUDAWrappers
{
    struct Simulation::Impl
    {
        Impl(int num_points, float domain_size, float theta, float dt) :
            points(num_points, domain_size),
            btree(num_points),
            octree(num_points, domain_size),
            bh(points.get_d_pos(), num_points, theta, dt),
            spawner(points.get_d_pos(), bh.get_d_vel(), num_points, 42),
            validator(points.get_d_pos(),
                      bh.get_d_vel(),
                      bh.get_d_acc(),
                      points.get_d_sort_indices_ptr(),
                      num_points,
                      dt,
                      5000) {}

        Impl(int num_points,
             float domain_size,
             float theta,
             float dt,
             void *mapped_ptrs[7]) :
            points(num_points,
                   domain_size,
                   static_cast<double *>(mapped_ptrs[0]),
                   static_cast<double *>(mapped_ptrs[1]),
                   static_cast<double *>(mapped_ptrs[2])),
            btree(num_points),
            octree(num_points,
                   domain_size,
                   static_cast<double *>(mapped_ptrs[3]),
                   static_cast<double *>(mapped_ptrs[4]),
                   static_cast<double *>(mapped_ptrs[5]),
                   static_cast<double *>(mapped_ptrs[6])),
            bh(points.get_d_pos(), num_points, theta, dt),
            spawner(points.get_d_pos(), bh.get_d_vel(), num_points, 42),
            validator(points.get_d_pos(),
                      bh.get_d_vel(),
                      bh.get_d_acc(),
                      points.get_d_sort_indices_ptr(),
                      num_points,
                      dt,
                      5000) {}

        void updatePoints()
        {
            points.compute_morton_codes();
            points.sort_by_codes();
            bh.sort_bodies(points.get_d_sort_indices_ptr());
            points.compute_unique_codes(btree.get_d_num_leaves_ptr());
            points.scan_attributes();
        }

        void updateOctree(int max_num_codes_per_leaf)
        {
            //std::cout << "num_unique_codes="
            //          << btree.get_num_leaves() << std::endl;
            btree.reset_max_num_leaves();
            btree.generate_leaves(points.get_d_unique_codes_ptr(),
                                  max_num_codes_per_leaf);
            _num_leaves = btree.get_num_leaves();
            btree.set_max_num_leaves(_num_leaves);
            //std::cout << "num_leaves=" << _num_leaves << std::endl;

            btree.build(points.get_d_unique_codes_ptr());
            btree.sort_to_bfs_order();
            btree.compute_octree_map();

            //btree.print();

            octree.set_max_num_nodes(btree.get_max_num_nodes());
            octree.build(btree);
            octree.compute_nodes_points_range(
                btree.get_d_leaf_first_code_idx_ptr(),
                points.get_d_codes_first_point_idx_ptr());
            octree.compute_nodes_barycenter(points);

            //octree.print();
        }

        void updateBodiesPos()
        {
            bh.solve_pos(octree,
                         points.get_d_codes_first_point_idx_ptr(),
                         btree.get_d_leaf_first_code_idx_ptr(),
                         _num_leaves);
        }

        void updateBodiesVel()
        {
            bh.solve_vel(octree,
                         points.get_d_codes_first_point_idx_ptr(),
                         btree.get_d_leaf_first_code_idx_ptr(),
                         _num_leaves);
        }

        Points<double> points;
        Btree btree;
        Octree<double> octree;
        BarnesHut<double> bh;
        Spawner<double> spawner;
        Validator<double> validator;

    private:
        int _num_leaves;
    };

    Simulation::Simulation(Simulation::Params &params) :
        _params(params)
    {
        _impl = std::make_unique<Simulation::Impl>(params.num_points,
                                                   params.domain_size,
                                                   params.theta,
                                                   params.dt);
    }

    Simulation::Simulation(Simulation::Params &params, GLuint buffers[7]) :
        _params(params)
    {
        void *mapped_ptrs[7];

        for (int i = 0; i < 7; ++i) {
            hipGraphicsResource_t res;
            // Mapping OpenGL buffer for access by CUDA
            hipGraphicsGLRegisterBuffer(&res,
                                         buffers[i],
                                         hipGraphicsRegisterFlagsNone);
            hipGraphicsMapResources(1, &res);
            // Obtaining device pointer
            size_t size;
            hipGraphicsResourceGetMappedPointer(&mapped_ptrs[i], &size, res);
        }

        _impl = std::make_unique<Simulation::Impl>(params.num_points,
                                                   params.domain_size,
                                                   params.theta,
                                                   params.dt,
                                                   mapped_ptrs);
    }

    void Simulation::spawnBodies()
    {
        _impl->spawner.sample_rotating_disk(0.3);
        //_impl->spawner.sample_uniform_pos();
        //_impl->spawner.sample_spherical_pos(0.3);
        //_impl->spawner.sample_plummer(0.1);

        //_impl->validator.copy_initial_conditions();
        _impl->updatePoints();
        _impl->updateOctree(_params.max_num_codes_per_leaf);
    }

    void Simulation::update()
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        //_impl->validator.update_all_pairs();

        // Solve for position
        _impl->updateBodiesPos();
        // Update octree
        _impl->updatePoints();
        _impl->updateOctree(_params.max_num_codes_per_leaf);
        // Solve for velocity
        _impl->updateBodiesVel();


        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);

        //std::cout << "elapsed=" << ms << std::endl;
    }

    int Simulation::get_num_octree_nodes()
    {
        return _impl->octree.get_num_nodes();
    }

    void Simulation::writeHistory(const std::string &csv_file_path)
    {
        _impl->validator.dump_history_to_csv(csv_file_path);
    }

    Simulation::~Simulation() {}
};
