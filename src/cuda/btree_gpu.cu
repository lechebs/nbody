#include "hip/hip_runtime.h"
#include "btree_gpu.cuh"

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/scatter.h>

__device__ __forceinline__ uint32_t _expand_bits(uint32_t u)
{
    u = (u * 0x00010001u) & 0xFF0000FFu;
    u = (u * 0x00000101u) & 0x0F00F00Fu;
    u = (u * 0x00000011u) & 0xC30C30C3u;
    u = (u * 0x00000005u) & 0x49249249u;

    return u;
}

// Computes 30-bit morton code by interleaving the bits
// of the coordinates, supposing that they are normalized
// in the range [0.0, 1.0]
__global__ void morton_encode(Points *points, uint32_t *codes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Scale coordinates to [0, 2^10)
    uint32_t x = (uint32_t) (points->get_x(idx) * 1023.0f);
    uint32_t y = (uint32_t) (points->get_y(idx) * 1023.0f);
    uint32_t z = (uint32_t) (points->get_z(idx) * 1023.0f);

    x = _expand_bits(x);
    y = _expand_bits(y);
    z = _expand_bits(z);

    // Left shift x by 2 bits, y by 1 bit, then bitwise or
    codes[idx] = x * 4 + y * 2 + z;
}

// Computes the longes common prefix between
// the bits of two unsigned integers
__device__ __forceinline__ int _lcp_safe(uint32_t *codes, int i, int j)
{
    // Does this allow coalescing?
    return __clz(codes[i] ^ codes[j]);
}

__device__ __forceinline__ int _lcp(uint32_t *codes, int i, int j, int n)
{
    // i index is always in range
if (j < 0 || j > n - 1)
        return -1;
    else 
        return _lcp_safe(codes, i, j);
}

__device__ __forceinline__ int _sign(int x)
{
    return (x > 0) - (x < 0);
}

__device__ int _find_split(uint32_t *codes,
                           int first,
                           int last,
                           int node_lcp,
                           int dir)
{
    int step = 0;
    int length = (last - first) * dir;

    do {
        length = (length + 1) >> 1;

        if (_lcp_safe(
                codes, first, first + (step + length) * dir) >
                node_lcp) {
            step += length;
        }
    } while (length > 1);

    return first + step * dir + min(dir, 0);
}

__global__ void build_radix_tree(uint32_t *codes, Btree *btree)
{
    int first = blockIdx.x * blockDim.x + threadIdx.x;

    int num_leaves = btree->get_num_leaves();
    if (first > num_leaves - 2) return;

    // Fill tmp arrays used for later computations
    btree->set_tmp_ranges(first, first);

    // Determines whether the left or right
    // leaf is part of the current internal node
    int d = _sign(_lcp(codes, first, first + 1, num_leaves) -
                  _lcp(codes, first, first - 1, num_leaves));

    // Minimum length of the common prefix between the
    // leaves covered by the current internal node, it
    // is obtained by computing lcp on the non-sibling
    // neighbouring node
    int lcp_min = _lcp(codes, first, first - d, num_leaves);

    // Computes upper bound for the length of prefix
    // covered by the current internal node by doubling
    // the search range until a leaf whose lcp is <= delta_min
    int max_length = 2;
    while (_lcp(codes, first, first + max_length * d, num_leaves) > lcp_min) {
        max_length = max_length << 1;
    }

    // Uses iterative binary search for the exact end of the
    // range of leaves covered by the current internal node
    int length = 0;
    int step = max_length;
    do {
        // Half the step size
        step = step >> 1;
        if (_lcp(codes, first, first + (length + step) * d, num_leaves) >
            lcp_min) {
            length += step;
        }
    } while (step > 1);
    // End of the range of leaves covered
    int last = first + length * d;

    // Length of prefix covered by the internal node
    int node_lcp = _lcp_safe(codes, first, last);
    int split = _find_split(codes, first, last, node_lcp, d);

    bool is_left_leaf = min(first, last) == split;
    bool is_right_left = max(first, last) == split + 1;

    // Record parent-child relationships
    btree->set_left(first, split, node_lcp, is_left_leaf);
    btree->set_right(first, split + 1, node_lcp, is_right_left);

    btree->set_depth(first, node_lcp / 3);
}

Btree::Btree(int num_leaves) : _num_leaves(num_leaves)
{
    // Allocating device memory for internal nodes
    hipMalloc(&_left, (num_leaves - 1) * sizeof(int));
    hipMalloc(&_right, (num_leaves - 1) * sizeof(int));
    hipMalloc(&_depth, (num_leaves - 1) * sizeof(int));
    // Allocating device memory for internal nodes and leaf nodes
    hipMalloc(&_edge_delta, (2 * num_leaves - 1) * sizeof(int));

    // Allocating device memory used for intermediate computations
    hipMalloc(&_tmp_range1, (num_leaves - 1) * sizeof(int));
    hipMalloc(&_tmp_range2, (num_leaves - 1) * sizeof(int));

    // Allocating object copy in device memory
    hipMalloc(&_d_this, sizeof(Btree));
    hipMemcpy(_d_this, this, sizeof(Btree), hipMemcpyHostToDevice);
}

void Btree::build(uint32_t *d_sorted_codes)
{
    build_radix_tree<<<_num_leaves / THREADS_PER_BLOCK,
                       THREADS_PER_BLOCK>>>(d_sorted_codes, _d_this);
}

void Btree::sort_to_bfs_order()
{
    // Sort arrays by depth
    thrust::stable_sort_by_key(
        thrust::device,
        _depth,
        _depth + _num_leaves - 1,
        // Consider using device_vector for all arrays
        thrust::make_zip_iterator(
            thrust::device_pointer_cast(_left),
            thrust::device_pointer_cast(_right),
            thrust::device_pointer_cast(_edge_delta),
            thrust::device_pointer_cast(_tmp_range1)));

    // Update child pointers

    // TODO: scatter_if _left1 and _right1 with condition
    // _left1[i] (_right1[i]) < num_leaves - 1 using as
    // map _tmp_range1 into _left2 and _right2, then copy_if
    // _left1 and _right1 into _left2 and _right2 using
    // the previous condition negated
}

Btree::~Btree()
{
    // Releasing device memory
    hipFree(_left);
    hipFree(_right);
    hipFree(_depth);
    hipFree(_edge_delta);

    hipFree(_d_this);
}


