#include "hip/hip_runtime.h"
#include "octree_gpu.cuh"

#include "btree_gpu.cuh"

#include <iostream>

// At most 15 nodes can be visited by traversing
// 3 levels of any subtree of the binary radix tree
#define _BUILD_STACK_SIZE 16

__global__ void _build_octree(Btree &btree, Octree &octree)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= btree.get_num_internal() ||
        !btree.is_octree_node(idx)) return;

    // TODO: can we avoid this?
    if (idx == 0) {
        octree.set_num_internal(btree.get_num_octree_nodes());
    }

    int parent = btree.get_octree_node(idx);

    octree.set_num_children(parent, 0);
    octree.set_leaves_range(parent,
                            btree.get_leaves_begin(idx),
                            btree.get_leaves_end(idx));

    // Stack used to traverse at most 3 levels
    int stack[_BUILD_STACK_SIZE];
    // Points to first and last+1 element on the stack
    int start = 0;
    int end = 0;
    stack[end++] = idx;

    do {
        int bin_node = stack[start++];
        int is_leaf = btree.is_leaf(bin_node);

        if (bin_node != idx &&
            (is_leaf || btree.is_octree_node(bin_node))) {

            int child = is_leaf ? btree.get_leaf(bin_node) :
                                  btree.get_octree_node(bin_node);

            octree.add_child(parent, child, is_leaf);

        } else {
            stack[end++] = btree.get_left(bin_node);
            stack[end++] = btree.get_right(bin_node);
        }

    } while (start != end);
}

Octree::Octree(int max_depth) : _max_depth(max_depth)
{
    _max_num_internal = 1;
    int num_nodes_at_depth = 1;
    // Nodes at max_depth are leaves
    for (int d = 0; d < max_depth - 1; ++d) {
        num_nodes_at_depth *= 8;
        _max_num_internal += num_nodes_at_depth;
    }

    // Allocating device memory to store octree nodes
    hipMalloc(&_children, _max_num_internal * 8 * sizeof(int));
    hipMalloc(&_num_children, _max_num_internal * sizeof(int));
    hipMalloc(&_leaves_begin, _max_num_internal * sizeof(int));
    hipMalloc(&_leaves_end, _max_num_internal * sizeof(int));

    // Allocating object copy in device memory
    hipMalloc(&_d_this, sizeof(Octree));
    hipMemcpy(_d_this, this, sizeof(Octree), hipMemcpyHostToDevice);
}

void Octree::build(Btree &btree)
{
    _build_octree<<<btree.get_num_internal() / THREADS_PER_BLOCK +
                    (btree.get_num_internal() % THREADS_PER_BLOCK > 0),
                    THREADS_PER_BLOCK>>>(*btree.get_dev_ptr(), *_d_this);
}

Octree::~Octree()
{
    hipFree(_children);
    hipFree(_num_children);
}
