#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/scan.h>
#include <thrust/gather.h>

//#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_merge_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>

#include "utils_gpu.cuh"
#include "btree_gpu.cuh"
#include "octree_gpu.cuh"

#define TIMER_START(start) hipEventRecord(start);
#define TIMER_STOP(msg, start, stop) {                   \
    hipEventRecord(stop);                               \
    hipEventSynchronize(stop);                          \
    float ms;                                            \
    hipEventElapsedTime(&ms, start, stop);              \
    std::cout << msg << ": " << ms << "ms" << std::endl; \
}

constexpr int NUM_POINTS = 2 << 8;
constexpr int MAX_CODES_PER_LEAF = 16;

void print_bits(uint32_t u)
{
    for (int i = 0; i < 32; ++i) {
        printf("%d", (u >> (31 - i)) & 0x01);
    }
}

int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    thrust::host_vector<float> h_x(NUM_POINTS);
    thrust::host_vector<float> h_y(NUM_POINTS);
    thrust::host_vector<float> h_z(NUM_POINTS);

    thrust::default_random_engine rng(100);
    thrust::uniform_real_distribution<float> dist;
    // thrust::random::normal_distribution<float> dist(0.5, 0.125);

    auto dist_gen = [&] { return max(0.0f, min(1.0f, dist(rng))); };

    thrust::generate(h_x.begin(), h_x.end(), dist_gen);
    thrust::generate(h_y.begin(), h_y.end(), dist_gen);
    thrust::generate(h_z.begin(), h_z.end(), dist_gen);

    // Allocate device memory to store points coordinates
    thrust::device_vector<float> d_x(h_x);
    thrust::device_vector<float> d_y(h_y);
    thrust::device_vector<float> d_z(h_z);
    float *d_x_ptr = thrust::raw_pointer_cast(&d_x[0]);
    float *d_y_ptr = thrust::raw_pointer_cast(&d_y[0]);
    float *d_z_ptr = thrust::raw_pointer_cast(&d_z[0]);

    // Initializes points SoA and copy to device
    Points h_points(d_x_ptr, d_y_ptr, d_z_ptr);
    Points *d_points = alloc_device_soa(&h_points, sizeof(Points));

    // Allocate device memory to store morton codes
    thrust::device_vector<uint32_t> d_codes(NUM_POINTS);
    uint32_t *d_codes_ptr = thrust::raw_pointer_cast(&d_codes[0]);

    // Kernel launch to compute morton codes of points
    morton_encode<<<NUM_POINTS / THREADS_PER_BLOCK,
                    THREADS_PER_BLOCK>>>(d_points, d_codes_ptr);

    LessOp custom_op;
    // Determine sort tmp storage size
    void *d_sort_tmp = nullptr;
    size_t sort_tmp_size;
    hipcub::DeviceMergeSort::SortPairs(d_sort_tmp,
                                    sort_tmp_size,
                                    d_codes_ptr,
                                    thrust::make_zip_iterator(d_x_ptr,
                                                              d_y_ptr,
                                                              d_z_ptr),
                                    NUM_POINTS,
                                    custom_op);
    // Allocate sort tmp storage
    hipMalloc(&d_sort_tmp, sort_tmp_size);
    // Sorting
    TIMER_START(start)
    // Faster than DeviceRadixSort::SortKeys and thrust::sort
    hipcub::DeviceMergeSort::SortPairs(d_sort_tmp,
                                    sort_tmp_size,
                                    d_codes_ptr,
                                    thrust::make_zip_iterator(d_x_ptr,
                                                              d_y_ptr,
                                                              d_z_ptr),
                                    NUM_POINTS,
                                    custom_op);
    TIMER_STOP("sort-codes", start, stop)
    hipFree(d_sort_tmp);

    // Allocating Btree for NUM_POINTS number of leaves,
    // the actual number of leaves will be smaller
    Btree h_btree(NUM_POINTS);

    // Obtaining unique codes and counting occurrences
    // using run-length encoding
    thrust::device_vector<uint32_t> d_unique_codes(NUM_POINTS);
    thrust::device_vector<int> d_codes_occurrences(NUM_POINTS);
    uint32_t *d_unique_codes_ptr =
        thrust::raw_pointer_cast(&d_unique_codes[0]);
    int *d_codes_occurrences_ptr =
        thrust::raw_pointer_cast(&d_codes_occurrences[0]);

    // WARNING: Only Btree device copy will store the actual number of leaves
    int *d_num_unique_codes = h_btree.get_dev_num_leaves_ptr();

    void *d_runlength_tmp = nullptr;
    size_t runlength_tmp_size;
    hipcub::DeviceRunLengthEncode::Encode(d_runlength_tmp,
                                       runlength_tmp_size,
                                       d_codes_ptr,
                                       d_unique_codes_ptr,
                                       d_codes_occurrences_ptr,
                                       d_num_unique_codes,
                                       NUM_POINTS);
    hipMalloc(&d_runlength_tmp, runlength_tmp_size);

    TIMER_START(start)
    hipcub::DeviceRunLengthEncode::Encode(d_runlength_tmp,
                                       runlength_tmp_size,
                                       d_codes_ptr,
                                       d_unique_codes_ptr,
                                       d_codes_occurrences_ptr,
                                       d_num_unique_codes,
                                       NUM_POINTS);
    TIMER_STOP("run-length", start, stop)
    hipFree(d_runlength_tmp);

    int h_num_unique_codes;
    hipMemcpy(&h_num_unique_codes,
               d_num_unique_codes,
               sizeof(int),
               hipMemcpyDeviceToHost);

    // Computing exclusive scan of d_codes_occurrences
    thrust::device_vector<int> d_scan_codes_occurrences(NUM_POINTS);
    int *d_scan_codes_occurrences_ptr =
        thrust::raw_pointer_cast(&d_scan_codes_occurrences[0]);

    void *d_scan_tmp = nullptr;
    size_t scan_tmp_size;
    hipcub::DeviceScan::ExclusiveSum(d_scan_tmp,
                                  scan_tmp_size,
                                  d_codes_occurrences_ptr,
                                  d_scan_codes_occurrences_ptr,
                                  NUM_POINTS);
    hipMalloc(&d_scan_tmp, scan_tmp_size);

    TIMER_START(start)
    hipcub::DeviceScan::ExclusiveSum(d_scan_tmp,
                                  scan_tmp_size,
                                  d_codes_occurrences_ptr,
                                  d_scan_codes_occurrences_ptr,
                                  NUM_POINTS);
    TIMER_STOP("codes-scan", start, stop)

    // We can use the same tmp storage to scan point coordinates
    // as long as we're dealing with 32 bit floats
    thrust::device_vector<float> d_scan_x(NUM_POINTS);
    thrust::device_vector<float> d_scan_y(NUM_POINTS);
    thrust::device_vector<float> d_scan_z(NUM_POINTS);
    float *d_scan_x_ptr = thrust::raw_pointer_cast(&d_scan_x[0]);
    float *d_scan_y_ptr = thrust::raw_pointer_cast(&d_scan_y[0]);
    float *d_scan_z_ptr = thrust::raw_pointer_cast(&d_scan_z[0]);

    TIMER_START(start)
    // TODO: parallelize over multiple streams?
    hipcub::DeviceScan::ExclusiveSum(d_scan_tmp,
                                  scan_tmp_size,
                                  d_x_ptr,
                                  d_scan_x_ptr,
                                  NUM_POINTS);
    hipcub::DeviceScan::ExclusiveSum(d_scan_tmp,
                                  scan_tmp_size,
                                  d_y_ptr,
                                  d_scan_y_ptr,
                                  NUM_POINTS);
    hipcub::DeviceScan::ExclusiveSum(d_scan_tmp,
                                  scan_tmp_size,
                                  d_z_ptr,
                                  d_scan_z_ptr,
                                  NUM_POINTS);
    TIMER_STOP("points-scan", start, stop)
    hipFree(d_scan_tmp);

    Points h_scan_points(d_scan_x_ptr, d_scan_y_ptr, d_scan_z_ptr);
    Points *d_scan_points = alloc_device_soa(&h_scan_points, sizeof(Points));

    // TODO: is it correct?
    // Octree h_octree(ceil(log2(num_unique_points) / 3) + 1)
    Octree h_octree(8);

    thrust::device_vector<int> d_leaf_first_code(NUM_POINTS + 1);
    int *d_leaf_first_code_ptr =
        thrust::raw_pointer_cast(&d_leaf_first_code[0]);

    TIMER_START(start)
    h_btree.generate_leaves(d_unique_codes_ptr,
                            d_leaf_first_code_ptr,
                            MAX_CODES_PER_LEAF);
    TIMER_STOP("btree-leaves", start, stop)

    /*
    thrust::host_vector<uint32_t> h_unique_codes(d_unique_codes);
    for (int i = 0; i < 32; ++i) {
        printf("%4d: %12u ", i, h_unique_codes[i]);
        print_bits(h_unique_codes[i]);
        printf("\n");
    }

    thrust::device_vector<int> h_leaf_first_code(d_leaf_first_code);
    for (int i = 0; i < NUM_POINTS + 1; ++i) {
        std::cout << "[" << i << "] " << h_leaf_first_code[i] << std::endl;
    }
    */

    TIMER_START(start)
    h_btree.build(d_unique_codes_ptr, d_leaf_first_code_ptr);
    TIMER_STOP("btree-build", start, stop)

    // WARNING: Perhaps sort octree instead?
    // Octree nodes are ~1/3 of the btree nodes,
    // sorting would be faster

    TIMER_START(start)
    h_btree.sort_to_bfs_order();
    TIMER_STOP("btree-sort", start, stop)

    TIMER_START(start)
    h_btree.compute_octree_map();
    TIMER_STOP("btree-scan", start, stop)

    TIMER_START(start)
    h_octree.build(h_btree);
    TIMER_STOP("octree-build", start, stop)

    TIMER_START(start)
    h_octree.compute_nodes_barycenter(d_points,
                                      d_scan_points,
                                      d_leaf_first_code_ptr,
                                      d_scan_codes_occurrences_ptr);
    TIMER_STOP("octree-barycenters", start, stop)

    // h_btree.print();
    h_octree.print();

    std::cout << "num_unique_codes=" << h_num_unique_codes << std::endl;

    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    hipFree(d_points);
    hipFree(d_scan_points);

    return 0;
}

